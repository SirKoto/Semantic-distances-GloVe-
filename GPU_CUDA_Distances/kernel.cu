#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <cstdint>
#include <stdio.h>
#include <vector>
#include <assert.h>
#include "GlobalHeader.h"
#define FULL_MASK 0xffffffff
#define N_THREADS 1024

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__device__ __constant__
embedV_t* c_model;
__device__ __constant__
embed_t* c_norms;

//rows determined as the amount of rows in a block
// A is query vector, B is the model ( rows ), C is output matrix
__global__ void DotProduct
(const int limit, const embed_t* A, embed_t* C, unsigned int* pos, const embed_t normA) {
	__shared__ embed_t fastA[numEmbeds];
	unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadIdx.x < numEmbeds) {
		fastA[threadIdx.x] = A[threadIdx.x]; // only one embeding is on A
	}
	__syncthreads();
	if (id < limit) {
		embed_t acum = 0;
		unsigned int row = id / 8; // Get row
		unsigned int interiorId = threadIdx.x % 8;  // Get id within row
		for (unsigned int i = interiorId; i < numEmbeds; i += 8) {
			acum += fastA[i] * c_model[row].data[i]; // Accumulate within the accumulator
		}        
        acum += __shfl_down_sync(FULL_MASK, acum, 4); // Reduction
        acum += __shfl_down_sync(FULL_MASK, acum, 2); // Reduction
        acum += __shfl_down_sync(FULL_MASK, acum, 1); // Reduction

		if (interiorId == 0) { // Final step and write results
			C[row] = acum / (normA * c_norms[row]);
			pos[row] = row;

		}
	}
}



__global__ void FirstMerge
(const int64_t N, embed_t *sims, unsigned int* pos, const int64_t length, const int64_t pad) {
	const int64_t id = blockIdx.x * blockDim.x + threadIdx.x;
	const int64_t start=id*N;
	const int64_t end=start+N;
    if (start<length) { 
    
    // Insertion sort, as N SHOULD be small
    
		for(int64_t i=start+1; i<end; i++)
		{
			if (i<length){
                /*if (i >= pad || i < 0) {
                    printf("ERRORR1 %i\n", i);
                }*/
				const embed_t temp=sims[i];
				const int64_t position=pos[i];
                int64_t j=i-1;
                
				while((j>=start) && (temp>sims[j]) )
				{
					sims[j+1]=sims[j];
					pos[j+1]=pos[j];
                    j=j-1;
                    /*if (j >= pad || j < -1) {
						printf("ERRORR3 %i\n", j);
					}*/
				}
				sims[(j+1)]=temp;
				pos[(j+1)]=position;
			}
			else if (i<pad) {
				for (int64_t i=0;i<N;++i) {
                    /*if (id+i >= pad || id+i < -1) {
						printf("ERRORR4 %i\n", i);
					}*/
    				sims[id+i]=0;
					pos[id+i]=0;
				}
			}
		}
	}
}

__global__ void BotchedMergeSort
(const int N, embed_t *sims, unsigned int* pos, const unsigned long stride) {

    unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    id=id*N;
	unsigned int posA=0,posB=0;
    if (id<stride) { 
		unsigned int buffPos[maxN];
		embed_t buffSims[maxN];
		
        embed_t elemA=sims[(id+stride)];
        unsigned int posAuxA=pos[(id+stride)];
        embed_t elemB=sims[id];
        unsigned int posAuxB=pos[id];

        for(unsigned int i=0;i<N;++i) {
            if (posAuxA==posAuxB) {
                ++posA;
                elemA=sims[(id+posA+stride)];
                posAuxA=pos[(id+posA+stride)];
            }
            if (elemA>elemB && posA<N) {
                ++posA;
                buffSims[i]=elemA;
                buffPos[i]=posAuxA;
                
                elemA=sims[(id+posA+stride)];
                posAuxA=pos[(id+posA+stride)];
            }
            else {
                ++posB;
                buffSims[i]=elemB;
                buffPos[i]=posAuxB;
                
                elemB=sims[id+posB];
                posAuxB=pos[id+posB];

            }
		}

		memcpy(sims + id, buffSims, N * sizeof(embed_t));
		memcpy(pos + id, buffPos, N * sizeof(unsigned int));
	
}
}


embed_t *A_d;
embed_t *C_d;
unsigned int *positions, *pos_d;


// FUNCTIONS DEFINED IN CUDAHELP.CU
extern "C"
void reservePinnedMemory(embed_t* &ptr, size_t bytes);

extern "C"
void freePinnedMemory(void* ptr);

// Load memory into cuda constants. This memory will be freed automatically at the end of the cuda context
extern "C"
void loadModel(embed_t * norms, embedV_t * model, uint32_t numRows, uint32_t N)
{
	assert(N <= maxN);

	fprintf(stdout, "Reserving memory for %i rows, and N %i\n", numRows, N);
	const size_t numBytesModel = sizeof(embedV_t) * numRows;
	const size_t numBytesNorms = sizeof(embed_t) * numRows;
	unsigned int numRowsMod=numRows;
    if (numRows%N!=0) numRowsMod=(N-numRows%N)+numRows;
	numRowsMod+=numRowsMod%2*N;
	const unsigned int numBytesQuery = sizeof(embedV_t);
	const unsigned int numBytesSims = sizeof(unsigned int) * numRowsMod;
	
	embedV_t* modelSym;
	embed_t* normsSym;

	gpuErrchk(hipMalloc((embed_t**)&modelSym, numBytesModel));
	gpuErrchk(hipMalloc((embed_t**)&normsSym, numBytesNorms));

	gpuErrchk(hipMemcpy(modelSym, model, numBytesModel, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(normsSym, norms, numBytesNorms, hipMemcpyHostToDevice));

	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(c_model), (void**)&modelSym, sizeof(modelSym)));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(c_norms), (void**)&normsSym, sizeof(normsSym)));

	    
	gpuErrchk(hipMalloc((embed_t**)&A_d, numBytesQuery));
	gpuErrchk(hipMalloc((embed_t**)&C_d, numBytesSims));
	gpuErrchk(hipMalloc((unsigned int**)&pos_d, numBytesSims));

	{
		embed_t* tmp;
		static_assert(sizeof(embed_t) == sizeof(unsigned int), "the embed type needs to be of 4 bytes");
		reservePinnedMemory(tmp, sizeof(embed_t) * numRowsMod);
		positions = reinterpret_cast<unsigned int*>(tmp);
	}

	gpuErrchk(hipDeviceSynchronize());// Comment this on release

}

extern "C"
void freeModel()
{
	gpuErrchk(hipFree(A_d));
	gpuErrchk(hipFree(C_d));
	gpuErrchk(hipFree(pos_d));
	freePinnedMemory(positions);

	embed_t* sym;
	gpuErrchk(hipMemcpyFromSymbol(&sym, HIP_SYMBOL(c_model), sizeof(embed_t*)));
	gpuErrchk(hipFree(sym));
	gpuErrchk(hipMemcpyFromSymbol(&sym, HIP_SYMBOL(c_norms), sizeof(embed_t*)));
	gpuErrchk(hipFree(sym));

}


// MAIN FUNCTION TO RUN

extern "C"
void runCuda(uint32_t numRows, embedV_t queryTerm, embed_t normA, uint32_t N, int &returnCode, std::vector<unsigned int> &res)
{

	assert(N <= maxN);


	unsigned int nBlocks=(numRows/128)+1;
    unsigned int nBlocksOriginal=nBlocks;
	float elapsedTime;
    
    unsigned int numRowsMod=numRows;
    if (numRows%N!=0) numRowsMod=(N-numRows%N)+numRows;
    numRowsMod+=numRowsMod%2*N;



	hipEvent_t start, stop;

	const unsigned int numBytesQuery = sizeof(embedV_t);
	// const unsigned int numBytesSims = sizeof(unsigned int) * numRowsMod;

	gpuErrchk(hipEventCreate(&start));
	gpuErrchk(hipEventCreate(&stop));


	gpuErrchk(hipMemcpyAsync(A_d, queryTerm.data, numBytesQuery, hipMemcpyHostToDevice));

	gpuErrchk(hipEventRecord(start, 0));

	DotProduct<<<nBlocks, N_THREADS >>>(numRows*8, A_d,  C_d, pos_d,normA);
    
	gpuErrchk(hipPeekAtLastError());
	//gpuErrchk(hipDeviceSynchronize());// Comment this on release
    
    FirstMerge<<<nBlocks, N_THREADS >>>(N,C_d,pos_d,numRows,numRowsMod);
	gpuErrchk(hipPeekAtLastError());
	//gpuErrchk(hipDeviceSynchronize());// Comment this on release

	unsigned long toReduce=((numRowsMod/N)/2);
	
    while(toReduce>0) {
        nBlocks=((toReduce*N)/ N_THREADS)+1;
		BotchedMergeSort <<<nBlocks, N_THREADS >>> (N, C_d, pos_d, toReduce * N);
		gpuErrchk(hipPeekAtLastError());
		//gpuErrchk(hipDeviceSynchronize()); // Comment this on release
        if (toReduce>1){
            toReduce+=toReduce%2;
		}
        toReduce=toReduce/2;
    }
	
	// Because we don't use the similarities rigt now...
	// gpuErrchk(hipMemcpyAsync(similarities, C_d, sizeof(embed_t)*N, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpyAsync(positions, pos_d, sizeof(unsigned int)*N, hipMemcpyDeviceToHost));

  hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
      fprintf(stderr, "ERROR: %s \n", hipGetErrorString(error));
      returnCode=1;
    }


	gpuErrchk(hipEventRecord(stop, 0));
	gpuErrchk(hipEventSynchronize(stop));
    
    

	gpuErrchk(hipEventElapsedTime(&elapsedTime, start, stop));
	printf("\nSimilarities\n");
	printf("Vector Size: %d\n", numRows);
	printf("nThreads: %d\n", N_THREADS);
	printf("nBlocks: %d\n", nBlocksOriginal+1);
	printf("Total Time with Events %4.6f ms\n", elapsedTime);
	printf("Bandwidth %4.3f GB/s\n", (numRows *numEmbeds* sizeof(float)) / (1000000 * elapsedTime));
  
	res.resize(N);
    for (unsigned int i=0;i<N;++i) {
		res[i] = positions[i];
    }
}


