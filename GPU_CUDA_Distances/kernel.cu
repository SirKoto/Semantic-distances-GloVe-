#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "GlobalHeader.h"
#include <vector>

#include <stdio.h>
#define CUDA_INCLUDE
#include "GlobalHeader.h"

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

//rows determined as the amount of rows in a block
// A is query vector, B is the model ( rows ), C is output matrix
// Rows should be 300 for proper usage of this access method
__global__ void DotProduct
(int rows, embed_t *A, embed_t *B, embed_t *C, embed_t normA, embed_t *normsB) {
  __shared__ embed_t fastA[300];
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id<300) {
      fastA[id]=A[id];
  }
  __syncthreads();
  embed_t acum=0;
  for(int i=0;i<300;++i) {
      acum+=fastA[i]*B[id*300+i];
  }
  C[id]=acum/(normA*normsB[id]);
}


__global__ void FirstMerge
(int N, float *sims, int length) {

  
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int start=id*N;
    int end=start+N;
    if (!(start>length)) { 
    
    // Insertion sort, as N SHOULD be small
   int key, j;
   for(int i = start+1; i<end; i++) {
      key = sims[i];
      j = i;
      while(j > 0 && sims[j-1]<key) {
         sims[j] = sims[j-1];
         j--;
      }
      sims[j] = key;  
   }
}
}

int binary_search(std::vector<std::string> words, int length, std::string to_be_found){
    
    int p = 0;
    int r = length - 1;
    int q = (r + p) / 2;
    int counter = 0;

    while (p <= r)
    {
        counter++;
        if (words[q] == to_be_found)
            return q;
        else
        {
            if (words[q] < to_be_found) 
            {
                p = q + 1;
                q = (r + p) / 2;
            }
            else
            {
                r = q - 1;
                q = (r + p) / 2;    
            }
        }
    }
    return -1;
}



extern "C"
int runCuda(std::vector<std::string> words,std::vector<embed_t> norms,std::vector<embedV_t> model,int rows,std::string toFind)
{

  embed_t* queryTerm,similarities;
  embedV_t* A_d,B_d;
  embed_t* C_d,norms_d;
  int nBlocks=1;
  int nThreads=10;
  float elapsedTime;

  embed_t similarities[300];

  hipEvent_t start,stop;

  int queryTermPos=binary_search(words,rows,toFind);
  for (int i=0;i<300;++i) {
      queryTerm[i]=model[queryTermPos*300+i];
  }
  
  embed_t normA=norms[queryTermPos];
  

  int numBytesQuery=sizeof(embed_t)*300;
  int numBytesModel=numBytesQuery*rows;
  int numBytesSimsAndNorms=sizeof(embed_t)*rows;

  hipEventCreate(&start);
  hipEventCreate(&stop);


  hipMalloc((embed_t**)&A_d, numBytesQuery); 
  hipMalloc((embed_t**)&B_d, numBytesModel); 
  hipMalloc((embed_t**)&C_d, numBytesSimsAndNorms); 
  hipMalloc((embed_t**)&norms_d, numBytesSimsAndNorms); 

  hipMemcpyAsync(A_d,queryTerm, numBytesQuery, hipMemcpyHostToDevice);
  hipMemcpyAsync(B_d,model, numBytesModel, hipMemcpyHostToDevice);
  hipMemcpyAsync(norms_d,norms, numBytesSimsAndNorms, hipMemcpyHostToDevice);

  hipEventRecord(start, 0);
  
  DotProduct<<<nBlocks,nThreads >>>(rows,A_d,B_d,C_d,normA,norms_d);

  hipMemcpyAsync(similarities, C_d, numBytesSimsAndNorms, hipMemcpyDeviceToHost); 
  
  hipFree(B_d);
  hipFree(norms_d);
  hipFree(A_d);

  
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);


  hipEventElapsedTime(&elapsedTime, start, stop);
  printf("\nSimilarities\n");
  printf("Vector Size: %d\n", rows);
  printf("nThreads: %d\n", nThreads);
  printf("nBlocks: %d\n", nBlocks);
  printf("Tiempo Total %4.6f ms\n", elapsedTime);
  printf("Ancho de Banda %4.3f GB/s\n", (rows *300* sizeof(float)) / (1000000 * elapsedTime));
  
  return 0;

}


