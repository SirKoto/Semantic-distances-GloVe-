#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <cstdint>
#include <stdio.h>
#include <vector>
#include "GlobalHeader.h"


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

//rows determined as the amount of rows in a block
// A is query vector, B is the model ( rows ), C is output matrix
// Rows should be 300 for proper usage of this access method
__global__ void DotProduct
(int rows, embed_t *A, embedV_t *B, embed_t *C,unsigned int *pos, embed_t normA, embed_t *normsB) {
  __shared__ embed_t fastA[numEmbeds];
  
  unsigned long id = blockIdx.x * blockDim.x + threadIdx.x;
  if (threadIdx.x<numEmbeds) {
      fastA[threadIdx.x]= A[threadIdx.x]; // only one embeding is on A
  }
  __syncthreads();
  if (id<rows) {
    
  //unsigned long identifier=id*numEmbeds;
  embed_t acum=0;
  for(unsigned long i=0;i<numEmbeds;++i) {
      acum+=fastA[i] * B[id].data[i];
  }
  C[id]=acum/(normA*normsB[id]);
  pos[id]=id;
  }
}



__global__ void FirstMerge
(int64_t N, embed_t *sims, unsigned int* pos, int64_t length, int64_t pad) {
	int64_t id = blockIdx.x * blockDim.x + threadIdx.x;
	int64_t start=id*N;
	int64_t end=start+N;
    if (start<length) { 
    
    // Insertion sort, as N SHOULD be small
    
		for(int64_t i=start+1; i<end; i++)
		{
			if (i<length){
                /*if (i >= pad || i < 0) {
                    printf("ERRORR1 %i\n", i);
                }*/
				embed_t temp=sims[i];
				int64_t position=pos[i];
                int64_t j=i-1;
                
				while((j>=start) && (temp>sims[j]) )
				{
					sims[j+1]=sims[j];
					pos[j+1]=pos[j];
                    j=j-1;
                    /*if (j >= pad || j < -1) {
						printf("ERRORR3 %i\n", j);
					}*/
				}
				sims[(j+1)]=temp;
				pos[(j+1)]=position;
			}
			else if (i<pad) {
				for (int64_t i=0;i<N;++i) {
                    /*if (id+i >= pad || id+i < -1) {
						printf("ERRORR4 %i\n", i);
					}*/
    				sims[id+i]=0;
					pos[id+i]=0;
				}
			}
		}
	}
}

__global__ void BotchedMergeSort
(int N, embed_t *sims,unsigned int* pos,embed_t *simsAux,unsigned int* posAux,unsigned long stride) {

  
    unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    id=id*N;
	unsigned int posA=0,posB=0;
    if (id<stride) { 
        embed_t elemA=sims[(id+stride)];
        unsigned int posAuxA=pos[(id+stride)];
        embed_t elemB=sims[id];
        unsigned int posAuxB=pos[id];

        sims[(id+stride)]=0;
        for(unsigned int i=0;i<N;++i) {
            if (posAuxA==posAuxB) {
                ++posA;
                elemA=sims[(id+posA+stride)];
                posAuxA=pos[(id+posA+stride)];
                sims[(id+posA+stride)]=0;
            }
            if (elemA>elemB && posA<N) {
                ++posA;
                simsAux[id+i]=elemA;
                posAux[id+i]=posAuxA;
                
                elemA=sims[(id+posA+stride)];
                posAuxA=pos[(id+posA+stride)];
                sims[(id+posA+stride)]=0;
            }
            else {
                ++posB;
                simsAux[id+i]=elemB;
                posAux[id+i]=posAuxB;
                
                elemB=sims[id+posB];
                posAuxB=pos[id+posB];

            }
        }
   
}
}


extern "C"
void loadModel(embed_t * norms, embedV_t * model, uint32_t numRows, embedV_t* &B_d, embed_t* & norms_d)
{
	size_t numBytesModel = sizeof(embedV_t) * numRows;
	size_t numBytesNorms = sizeof(embed_t) * numRows;

	gpuErrchk(hipMalloc((embed_t**)&B_d, numBytesModel));
	gpuErrchk(hipMalloc((embed_t**)&norms_d, numBytesNorms));

	gpuErrchk(hipMemcpy(B_d, model, numBytesModel, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(norms_d, norms, numBytesNorms, hipMemcpyHostToDevice));
}

extern "C"
void freeAll(embedV_t * &B_d, embed_t * &norms_d)
{
	gpuErrchk(hipFree(norms_d));
	gpuErrchk(hipFree(B_d));
}


extern "C"
void runCuda(embed_t* norms, embedV_t* model, uint32_t numRows, uint32_t queryTermPos, uint32_t N, embedV_t * B_d, embed_t * norms_d, int &returnCode, std::vector<unsigned int> &res)
{
	if (!B_d || !norms_d) {
		fprintf(stderr, "Memory not initialized\n");
		returnCode = 1;
		res = {};
		return;
	}

	embedV_t queryTerm;
	embed_t* A_d;
    embed_t *C_d,*CAux_d;
    unsigned int *positions,*pos_d,*posAux_d;
	unsigned int nBlocks=(numRows/512)+1;
	int nThreads=512;
	float elapsedTime;
    
    unsigned int numRowsMod=numRows;
    if (numRows%N!=0) numRowsMod=(N-numRows%N)+numRows;
    numRowsMod+=numRowsMod%2*N;

    //printf("%u\n",numRows);
	embed_t* similarities;
	gpuErrchk(hipHostMalloc((void**)&similarities, sizeof(embed_t) * numRowsMod));
	gpuErrchk(hipHostMalloc((void**)&positions, sizeof(embed_t) * numRowsMod));


	hipEvent_t start, stop;

	queryTerm = model[queryTermPos]; // request the model to look for
    
	embed_t normA = norms[queryTermPos];


	unsigned int numBytesQuery = sizeof(embedV_t);
	unsigned int numBytesSims = sizeof(unsigned int) * numRowsMod;

	gpuErrchk(hipEventCreate(&start));
	gpuErrchk(hipEventCreate(&stop));
    
	gpuErrchk(hipMalloc((embed_t**)&A_d, numBytesQuery));
	gpuErrchk(hipMalloc((embed_t**)&C_d, numBytesSims));
	gpuErrchk(hipMalloc((unsigned int**)&pos_d, numBytesSims));


	gpuErrchk(hipMalloc((embed_t**)&CAux_d, numBytesSims));
	gpuErrchk(hipMalloc((unsigned int**)&posAux_d, numBytesSims));


	gpuErrchk(hipMemcpyAsync(A_d, queryTerm.data, numBytesQuery, hipMemcpyHostToDevice));

	gpuErrchk(hipEventRecord(start, 0));

	DotProduct<<<nBlocks, nThreads >>>(numRows, A_d, B_d, C_d, pos_d,normA, norms_d);
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());// Coment this on release
    
    FirstMerge<<<nBlocks, nThreads >>>(N,C_d,pos_d,numRows,numRowsMod);
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());// Coment this on release

    unsigned long toReduce=((numRowsMod/N)/2);
    bool alternate=true;
    while(toReduce>0) {
        nBlocks=((toReduce*N)/nThreads)+1;
        //printf("%lu\n",toReduce*N);
		if (alternate) { 
			BotchedMergeSort <<<nBlocks, nThreads >>> (N, C_d, pos_d, CAux_d, posAux_d, toReduce * N); 
			gpuErrchk(hipPeekAtLastError());
			gpuErrchk(hipDeviceSynchronize());// Coment this on release
		}
		else {
			BotchedMergeSort <<<nBlocks, nThreads >>> (N, CAux_d, posAux_d, C_d, pos_d, toReduce * N);
			gpuErrchk(hipPeekAtLastError());
			gpuErrchk(hipDeviceSynchronize()); // Coment this on release
		}
        if (toReduce>1){
            toReduce+=toReduce%2;
            }
        toReduce=toReduce/2;
        alternate=!alternate;
    }
    
    if (alternate) {
		gpuErrchk(hipMemcpyAsync(similarities, C_d, sizeof(embed_t)*N, hipMemcpyDeviceToHost));
		gpuErrchk(hipMemcpyAsync(positions, pos_d, sizeof(unsigned int)*N, hipMemcpyDeviceToHost));
    }
    else {
		gpuErrchk(hipMemcpyAsync(similarities, CAux_d, sizeof(embed_t)*N, hipMemcpyDeviceToHost));
		gpuErrchk(hipMemcpyAsync(positions, posAux_d, sizeof(unsigned int)*N, hipMemcpyDeviceToHost));
    }

  hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
      fprintf(stderr, "ERROR: %s \n", hipGetErrorString(error));
      returnCode=1;
    }

	gpuErrchk(hipFree(A_d));
	gpuErrchk(hipFree(CAux_d));
	gpuErrchk(hipFree(C_d));
	gpuErrchk(hipFree(pos_d));
	gpuErrchk(hipFree(posAux_d));

	gpuErrchk(hipEventRecord(stop, 0));
	gpuErrchk(hipEventSynchronize(stop));
	
    //printf("\nSimilarity vector\n");
    
   /*for(int i=0;i<N;++i) {
    printf("[ %f , %i ]",similarities[i],positions[i]);

    }*/
    
    

	gpuErrchk(hipEventElapsedTime(&elapsedTime, start, stop));
	printf("\nSimilarities\n");
	printf("Vector Size: %d\n", numRows);
	printf("nThreads: %d\n", nThreads);
	printf("nBlocks: %d\n", (numRows/nThreads)+1);
	printf("Tiempo Total %4.6f ms\n", elapsedTime);
	printf("Ancho de Banda %4.3f GB/s\n", (numRows *numEmbeds* sizeof(float)) / (1000000 * elapsedTime));
  
    std::vector<unsigned int> results;
    for (unsigned int i=0;i<N;++i) {
		results.push_back(positions[i]);
    }

	gpuErrchk(hipHostFree(similarities));
	gpuErrchk(hipHostFree(positions));

	res = results;
}


