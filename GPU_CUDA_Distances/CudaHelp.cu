#include "hip/hip_runtime.h"
#include ""
#include <cstdint>
#include <stdio.h>
#include "GlobalHeader.h"
#include <assert.h>


// Reserve pinned memory
extern "C"
void reservePinnedMemory(embed_t* &ptr, size_t bytes) {

    hipError_t status = hipHostMalloc((void**)&ptr, bytes);
	if (status != hipSuccess)
	{
		fprintf(stderr, "CUDA Runtime Error: %s\n",
			hipGetErrorString(status));

		assert(status == hipSuccess);
	}
}

extern "C"
void reservePinnedMemoryV(embedV_t * &ptr, size_t bytes) {
    
	hipError_t status = hipHostMalloc((void**)&ptr, bytes);
	if (status != hipSuccess)
	{
		fprintf(stderr, "CUDA Runtime Error: %s\n",
			hipGetErrorString(status));

		assert(status == hipSuccess);
	}
}


// Free all data from pinned
extern "C"
void freePinnedMemory(void* ptr) {
	hipHostFree(ptr);
}
