#include "hip/hip_runtime.h"
#include ""
#include <cstdint>
#include <stdio.h>
#include "GlobalHeader.h"
#include <assert.h>


// Reserve pinned memory
extern "C"
void reservePinnedMemory(embed_t* &ptr, size_t bytes) {

	#ifdef NOT_PINNED_MEMORY
	ptr = (embed_t *) malloc(bytes);
	#else
	hipError_t status = hipHostMalloc((void**)&ptr, bytes);
	if (status != hipSuccess)
	{
		fprintf(stderr, "CUDA Runtime Error: %s\n",
			hipGetErrorString(status));

		assert(status == hipSuccess);
	}
	#endif
}

extern "C"
void reservePinnedMemoryV(embedV_t * &ptr, size_t bytes) {
	#ifdef NOT_PINNED_MEMORY
	ptr = (embedV_t *) malloc(bytes);
	#else
	hipError_t status = hipHostMalloc((void**)&ptr, bytes);
	if (status != hipSuccess)
	{
		fprintf(stderr, "CUDA Runtime Error: %s\n",
			hipGetErrorString(status));

		assert(status == hipSuccess);
	}
	#endif
}


// Free all data from pinned
extern "C"
void freePinnedMemory(void* ptr) {
	#ifdef NOT_PINNED_MEMORY
	free(ptr);
	#else
	hipHostFree(ptr);
	#endif
}
